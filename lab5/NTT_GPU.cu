#include <cstring>
#include <string>
#include <iostream>
#include <fstream>
#include <chrono>
#include <iomanip>
#include <sys/time.h>
#include <hip/hip_runtime.h>

typedef long long LL;
const int G = 3;

// ---------------- 输入输出函数保持不变 ----------------
void fRead(int *a, int *b, int *n, int *p, int input_id) {
    std::string str1 = "../data/";
    std::string str2 = std::to_string(input_id);
    std::string strin = str1 + str2 + ".in";
    char data_path[strin.size() + 1];
    std::copy(strin.begin(), strin.end(), data_path);
    data_path[strin.size()] = '\0';
    std::ifstream fin;
    fin.open(data_path, std::ios::in);
    fin >> *n >> *p;
    for (int i = 0; i < *n; i++) fin >> a[i];
    for (int i = 0; i < *n; i++) fin >> b[i];
}

void fCheck(int *ab, int n, int input_id) {
    std::string str1 = "../data/";
    std::string str2 = std::to_string(input_id);
    std::string strout = str1 + str2 + ".out";
    char data_path[strout.size() + 1];
    std::copy(strout.begin(), strout.end(), data_path);
    data_path[strout.size()] = '\0';
    std::ifstream fin;
    fin.open(data_path, std::ios::in);
    for (int i = 0; i < n * 2 - 1; i++) {
        int x; fin >> x;
        if (x != ab[i]) {
            std::cout << "多项式乘法结果错误" << std::endl;
            return;
        }
    }
    std::cout << "多项式乘法结果正确" << std::endl;
}

void fWrite(int *ab, int n, int input_id) {
    std::string str1 = "../files/";
    std::string str2 = std::to_string(input_id);
    std::string strout = str1 + str2 + ".out";
    char output_path[strout.size() + 1];
    std::copy(strout.begin(), strout.end(), output_path);
    output_path[strout.size()] = '\0';
    std::ofstream fout;
    fout.open(output_path, std::ios::out);
    for (int i = 0; i < n * 2 - 1; i++) fout << ab[i] << '\n';
}

// ---------------- 工具函数 ----------------
__host__ __device__ inline int qpow(int x, int y, int p) {
    int ret = 1;
    while (y) {
        if (y & 1) ret = (LL)ret * x % p;
        x = (LL)x * x % p;
        y >>= 1;
    }
    return ret;
}

__host__ int getinv(int x, int p) {
    int a = x, b = p, u = 1, v = 0;
    while (b) {
        int t = a / b;
        a -= t * b; std::swap(a, b);
        u -= t * v; std::swap(u, v);
    }
    return (u + p) % p;
}

__global__ void bit_reverse_copy(int *a, int *rev, int len) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < len && tid < rev[tid]) {
        int tmp = a[tid];
        a[tid] = a[rev[tid]];
        a[rev[tid]] = tmp;
    }
}

__global__ void ntt_stage(int *a, int len, int step, int *roots, int p) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int m = step << 1;
    int start = (tid / step) * m + (tid % step);
    if (start + step < len) {
        int u = a[start];
        int v = (LL)a[start + step] * roots[step + (tid % step)] % p;
        a[start] = (u + v) % p;
        a[start + step] = (u - v + p) % p;
    }
}

__global__ void pointwise_mul(int *a, int *b, int *c, int len, int p) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < len) c[i] = (LL)a[i] * b[i] % p;
}

void poly_multiply_cuda(int *a, int *b, int *ab, int n, int p) {
    int len = 1, loglen = -1;
    while (len < 2 * n - 1) len <<= 1, loglen++;

    int *ta = new int[len]();
    int *tb = new int[len]();
    memcpy(ta, a, n * sizeof(int));
    memcpy(tb, b, n * sizeof(int));

    int *rev = new int[len];
    for (int i = 0; i < len; i++)
        rev[i] = (rev[i >> 1] >> 1) | ((i & 1) << loglen);

    int *root = new int[len<<1];
    for (int i = 1; i < len; i <<= 1) {
        int wn = qpow(G, (p - 1) / (i << 1), p);
        root[i] = 1;
        for (int j = 1; j < i; j++)
            root[i + j] = (LL)root[i + j - 1] * wn % p;
    }

    int Ginv = getinv(G, p);
    int *inv_root = new int[len << 1];
    for (int i = 1; i < len; i <<= 1) {
        int wn = qpow(Ginv, (p - 1) / (i << 1), p);
        inv_root[i] = 1;
        for (int j = 1; j < i; j++)
            inv_root[i + j] = (LL)inv_root[i + j - 1] * wn % p;
    }

    int *da, *db, *dc, *drev, *droot, *d_inv_root;
    hipMalloc(&da, len * sizeof(int));
    hipMalloc(&db, len * sizeof(int));
    hipMalloc(&dc, len * sizeof(int));
    hipMalloc(&drev, len * sizeof(int));
    hipMalloc(&droot, (len<<1) * sizeof(int));
    hipMalloc(&d_inv_root, (len << 1) * sizeof(int));

    hipMemcpy(da, ta, len * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(db, tb, len * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(drev, rev, len * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(droot, root, (len<<1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_inv_root, inv_root, (len << 1) * sizeof(int), hipMemcpyHostToDevice);


    int block = 512, grid = (len + block - 1) / block;

    bit_reverse_copy<<<grid, block>>>(da, drev, len);
    for (int s = 1; s < len; s <<= 1)
        ntt_stage<<<grid, block>>>(da, len, s, droot, p);

    bit_reverse_copy<<<grid, block>>>(db, drev, len);
    for (int s = 1; s < len; s <<= 1)
        ntt_stage<<<grid, block>>>(db, len, s, droot, p);

    pointwise_mul<<<grid, block>>>(da, db, dc, len, p);

    bit_reverse_copy<<<grid, block>>>(dc, drev, len);
    for (int s = 1; s < len; s <<= 1)
        ntt_stage<<<grid, block>>>(dc, len, s, d_inv_root, p);

    int inv_len = getinv(len, p);
    hipMemcpy(ab, dc, len * sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < len; i++)
        ab[i] = (LL)ab[i] * inv_len % p;

    delete[] ta; delete[] tb; delete[] rev; delete[] root; delete[] inv_root;
    hipFree(da); hipFree(db); hipFree(dc); hipFree(drev); hipFree(droot); hipFree(d_inv_root);
}

int a[300000], b[300000], ab[300000];

int main() {
    int test_begin = 0, test_end = 3;
    for(int i = test_begin; i <= test_end; ++i){
        std::cout << "test id: " << i << "\n";
        long double ans = 0;
        int n_, p_;
        memset(a, 0, sizeof(a));
        memset(b, 0, sizeof(b));
        fRead(a, b, &n_, &p_, i);
        memset(ab, 0, sizeof(ab));
        auto Start = std::chrono::high_resolution_clock::now();
        poly_multiply_cuda(a, b, ab, n_, p_);
        hipDeviceSynchronize();  // 等待 GPU 计算完成
        auto End = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double,std::ratio<1,1000>>elapsed = End - Start;
        ans += elapsed.count();
        fCheck(ab, n_, i);
        std::cout<<"average latency for n = "<<n_<<" p = "<<p_<<" : "<<ans<<" (ms) "<<std::endl;
        fWrite(ab, n_, i);
    }
    return 0;
}